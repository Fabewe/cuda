
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>

using namespace std;

__global__ void identifyThread(){
    int i = threadIdx.x;
    printf("Soy el hilo #%d\n",i);
}

int main(int argc, char ** argv){
    identifyThread<<<1,stoi(argv[1])>>>();
}