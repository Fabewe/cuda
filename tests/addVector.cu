
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>

#define N 1000

__global__ void sumaVec(float * out, float * a,float *b, int size){
    for (int i = 0;i<size;i++){
        out[i] = a[i] + b[i];
    }
    
}

int main(int argc,char ** argv) {

    clock_t t;
    t = clock();
    float *sum1,*sum2,*out;
    float *c_sum1,*c_sum2,*c_out;


    sum1 = (float *)malloc(sizeof(float)*N);
    sum2 = (float *)malloc(sizeof(float)*N);
    out = (float *)malloc(sizeof(float)*N);


    for (int i = 0 ;i<N;i++){
        sum1[i] = 1.0;
        sum2[i] = 2.0;
    }


    hipMalloc((void**)&c_sum1,sizeof(float)*N);
    hipMalloc((void**)&c_sum2,sizeof(float)*N);
    hipMalloc((void**)&c_out,sizeof(float)*N);

    hipMemcpy(c_sum1,sum1,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(c_sum2,sum2,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(c_sum2,sum2,sizeof(float)*N,hipMemcpyHostToDevice);
 

    sumaVec<<<std::stoi(argv[1]),std::stoi(argv[2])>>>(c_out,c_sum1,c_sum2,N);

    hipMemcpy(out,c_out,sizeof(float)*N,hipMemcpyDeviceToHost);


    t = clock() - t;

    for(int i = 0;i< N;i++){
            printf("out[%d] = %f\n",i,out[i]);
        }
    hipFree(c_sum1);
    hipFree(c_sum2);
    hipFree(c_out);


    free(sum1);
    free(sum2);
    free(out);

    printf ("Tiempo = %f seconds.\n",((float)t)/CLOCKS_PER_SEC);


    return 0;
}
