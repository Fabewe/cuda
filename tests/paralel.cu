
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>

#define N 5

__global__ void sumaVec(float * out, float * a,float *b, int size){

    if(threadIdx.x < size){
        out[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
    }
}

int main(int argc,char ** argv) {

    clock_t t;
    t = clock();
    float *sum1,*sum2,*out;
    float *c_sum1,*c_sum2,*c_out;


    sum1 = (float *)malloc(sizeof(float)*N);
    sum2 = (float *)malloc(sizeof(float)*N);
    out = (float *)malloc(sizeof(float)*N);


    for (int i = 0 ;i<N;i++){
        sum1[i] = 1.0;
        sum2[i] = 2.0;
    }


    hipMalloc((void**)&c_sum1,sizeof(float)*N);
    hipMalloc((void**)&c_sum2,sizeof(float)*N);
    hipMalloc((void**)&c_out,sizeof(float)*N);

    hipMemcpy(c_sum1,sum1,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(c_sum2,sum2,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(c_sum2,sum2,sizeof(float)*N,hipMemcpyHostToDevice);
 
    int numCud = 0;
    do{
        numCud++;
    }while(numCud*32<N);

    printf("NumCud = %d\n",numCud);

    sumaVec<<<1,32*numCud>>>(c_out,c_sum1,c_sum2,N);

    hipMemcpy(out,c_out,sizeof(float)*N,hipMemcpyDeviceToHost);


    t = clock() - t;

    for(int i = 0;i< N;i++){
            printf("out[%d] = %f\n",i,out[i]);
    }
    hipFree(c_sum1);
    hipFree(c_sum2);
    hipFree(c_out);


    free(sum1);
    free(sum2);
    free(out);

    printf ("Tiempo = %f seconds.\n",((float)t)/CLOCKS_PER_SEC);


    return 0;
}